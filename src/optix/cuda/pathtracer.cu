#include "hip/hip_runtime.h"
#include "common/types.cuh"
#include "common/helper_math.cuh"

static __forceinline__ __device__
void *unpackPointer( uint32_t i0, uint32_t i1 )
{
const uint64_t uptr = static_cast<uint64_t>( i0 ) << 32 | i1;
void*           ptr = reinterpret_cast<void*>( uptr ); 
return ptr;
}

static __forceinline__ __device__
void  packPointer( void* ptr, uint32_t& i0, uint32_t& i1 )
{
const uint64_t uptr = reinterpret_cast<uint64_t>( ptr );
i0 = uptr >> 32;
i1 = uptr & 0x00000000ffffffff;
}

template<typename T>
static __forceinline__ __device__ T *getPRD()
{ 
const uint32_t u0 = optixGetPayload_0();
const uint32_t u1 = optixGetPayload_1();
return reinterpret_cast<T*>( unpackPointer( u0, u1 ) );
}

extern "C" {

__constant__ fart::LaunchParams launch_params;


__global__ void __miss__pathtrace() {
    float3 &prd = *(float3*)getPRD<float3>();
    // set to constant white as background color
    prd = make_float3(1.f, 1.f, 1.f);
}

__global__ void __closesthit__pathtrace() {
    float3 &prd = *(float3*)getPRD<float3>();
    // set to constant white as background color
    prd = make_float3(optixGetTriangleBarycentrics().x, optixGetTriangleBarycentrics().y, 0.f);
}

__global__ void __raygen__pathtrace() {
    const int ix = optixGetLaunchIndex().x;
    const int iy = optixGetLaunchIndex().y;
    const int pid = iy * launch_params.viewport_size.x + ix;
    launch_params.framebuffer_rgba[pid].x = (float) ix / launch_params.viewport_size.x;
    launch_params.framebuffer_rgba[pid].y = (float) iy / launch_params.viewport_size.y;
    launch_params.framebuffer_rgba[pid].z = 1.f;
    launch_params.framebuffer_rgba[pid].a = 1.f;

    float3 pixelColorPRD;

    // the values we store the PRD pointer in:
    uint32_t u0, u1;
    packPointer( &pixelColorPRD, u0, u1 );

    float2 screen = make_float2(((float)ix+0.5f)/launch_params.viewport_size.x, 
                                ((float)iy+0.5f)/launch_params.viewport_size.y);

    // generate ray direction
    float3 rayDir = make_float3(launch_params.camera.dir.x, launch_params.camera.dir.y, launch_params.camera.dir.z);
    float3 camUp = make_float3(launch_params.camera.up.x, launch_params.camera.up.y, launch_params.camera.up.z);
    float3 camRight = cross(camUp, rayDir);

    rayDir = normalize(rayDir
                        + (screen.x - 0.5f) * camRight * launch_params.aspect_ratio
                        + (screen.y - 0.5f) * camUp);

    float3 rayOrg;
    rayOrg.x = launch_params.camera.eye.x;
    rayOrg.y = launch_params.camera.eye.y;
    rayOrg.z = launch_params.camera.eye.z;

    optixTrace(launch_params.traversable,
               rayOrg,
               rayDir,
               0.f,    // tmin
               1e20f,  // tmax
               0.0f,   // rayTime
               OptixVisibilityMask( 255 ),
               OPTIX_RAY_FLAG_DISABLE_ANYHIT,//OPTIX_RAY_FLAG_NONE,
               0,             // SBT offset
               1,             // SBT stride
               0,             // missSBTIndex 
               u0, u1 );
            
    launch_params.framebuffer_rgba[pid].x = pixelColorPRD.x;
    launch_params.framebuffer_rgba[pid].y = pixelColorPRD.y;
    launch_params.framebuffer_rgba[pid].z = pixelColorPRD.z;
    launch_params.framebuffer_rgba[pid].a = 1.f;
}

}
